#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4,  sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool use_robust_method>
static __device__
void intersect_sphere(void)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float3 D = ray.direction; // unit vector of ray direction
  float radius = sphere.w;

  float a = dot(D,D);
  float b = 2.0f*dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-4.0f * a * c; // delta
  if(disc >= 0.0f)
    {
      float sdisc = sqrtf(disc);
      float denom = 2.0f * a;
      float root1 = (-b+sdisc)/denom;
      float root2 = (-b-sdisc)/denom;
      float t_min = fminf(root1,root2);
      float t_max = fmaxf(root1,root2);
        

      if (t_min<=t_max) 
      {
        bool check_second =true;
        if (rtPotentialIntersection(t_min))
        {
          shading_normal=geometric_normal= normalize(O + t_min*D);
          if(rtReportIntersection(0)) 
            check_second=false;
        }

        if (check_second)
        {
          if(rtPotentialIntersection(t_max))
          {
            shading_normal=geometric_normal=normalize(O + t_max*D);
            rtReportIntersection(0);
          }
        }
      }
    }
}


RT_PROGRAM void intersect(int primIdx)
{
  intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_sphere<true>();
}


RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

