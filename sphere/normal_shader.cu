#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "per_ray_data.h"
#include "common.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_color, , );
rtBuffer<BasicLight> light_buffer;


RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0.5f,0.3f,0.9f);

  rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,shading_normal));
	float3 ffnormal = optix::faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_color;
	float3 hit_point=ray.origin + t_hit * ray.direction;

	for(int i = 0; i < light_buffer.size(); ++i)
	{
		BasicLight light = light_buffer[i];
		float3 L=normalize(light.pos - hit_point);
		float Diffuse_index=dot(L,ffnormal);

		if (Diffuse_index >= 0) color += Kd * Diffuse_index * light.color;
	}

  prd_radiance.result = color;
}
